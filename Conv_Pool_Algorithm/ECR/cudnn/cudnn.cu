#include <iostream>
#include <stdio.h>
#include <malloc.h>
#include <fstream>

#include <assert.h>


// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cudnn.h>

using namespace std;



int main() {
  

  // convolution
  const int pad_h = 0;
  const int pad_w = 0;
  const int str_h = 1;
  const int str_w = 1;
  const int dil_h = 1;
  const int dil_w = 1;
  
	string a[4995];
	int h[4995],w[4995];
	ifstream file_list("../../dataset/file_list");
	for(int i=0;i<4995;i++)
		file_list>>a[i];
	file_list.close();
	ifstream conv_shape("../../dataset/conv_shape");
	for(int i=0;i<4995;i++){
		conv_shape>>h[i];
		w[i] = h[i];
	}
	conv_shape.close();
//	printf("read success!\n");
	
	// per-experiment
	int len;
	
	
	 // filter
	const int filt_k = 1;
	const int filt_c = 1;
	const int filt_h = 3;
	const int filt_w = 3;
	
	const int kernelSize = filt_w*filt_h;
 	const float kernel[kernelSize] = { 1,0,1,0,1,1,0,1,1};
  
	 
	 for(int i=0;i<4995;i++){
	 
   // input
   const int in_n = 1;
   const int in_c = 1;
    int in_h = h[i];
    int in_w = w[i];

	int arraySize = in_h*in_w;
	//int o_w = (i_w-k_w)/stride +1;
	//int o_h = (i_h-k_h)/stride +1;
	
			
		
	float *feature = new float[arraySize];
	len = 0;
	
	ifstream conv_feature(("../../dataset/conv/"+a[i]).c_str());
	while(!conv_feature.eof())
		conv_feature>>feature[len++];
	conv_feature.close();	 
  	//printf("feature read success!\n");
	// output
	int out_n;
	int out_c;
	int out_h;
	int out_w;
  
      hipEvent_t start,stop;
      float elapsedTime1 = 0.0;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start,0);
	cudnnHandle_t cudnn;
   (cudnnCreate(&cudnn));
  cudnnTensorDescriptor_t in_desc;
   (cudnnCreateTensorDescriptor(&in_desc));
   (cudnnSetTensor4dDescriptor(
        in_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        in_n, in_c, in_h, in_w));

  float *in_data;
   (hipMalloc(
        &in_data, in_n * in_c * in_h * in_w * sizeof(float)));

 


  cudnnFilterDescriptor_t filt_desc;
   (cudnnCreateFilterDescriptor(&filt_desc));
   (cudnnSetFilter4dDescriptor(
        filt_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));

  float *filt_data;
   (hipMalloc(
      &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));




  cudnnConvolutionDescriptor_t conv_desc;
   (cudnnCreateConvolutionDescriptor(&conv_desc));
   (cudnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        CUDNN_CONVOLUTION, CUDNN_DATA_FLOAT));


  
   (cudnnGetConvolution2dForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        &out_n, &out_c, &out_h, &out_w));



  cudnnTensorDescriptor_t out_desc;
   (cudnnCreateTensorDescriptor(&out_desc));
   (cudnnSetTensor4dDescriptor(
        out_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

  float *out_data;
   (hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

  // algorithm
  const int n_requestedAlgo = 20;
  cudnnConvolutionFwdAlgoPerf_t algo_perf[n_requestedAlgo];
  int n_returnedAlgo;
  // = CUDNN_CONVOLUTION_FWD_PREFER_FASTEST; // CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
 (cudnnFindConvolutionForwardAlgorithm(
        cudnn,
        in_desc, filt_desc, conv_desc, out_desc,
        n_requestedAlgo, &n_returnedAlgo, algo_perf));

  cudnnConvolutionFwdAlgo_t algo = algo_perf[0].algo;

  // workspace
  size_t ws_size;
   (cudnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

  float *ws_data;
   (hipMalloc(&ws_data, ws_size));



  // perform
  float alpha = 1.f;
  float beta = 0.f;
 
	hipMemcpy(in_data,feature,arraySize*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(filt_data,kernel,9*sizeof(float),hipMemcpyHostToDevice);
      
       (cudnnConvolutionForward(
      cudnn,
      &alpha, in_desc, in_data, filt_desc, filt_data,
      conv_desc, algo, ws_data, ws_size,
      &beta, out_desc, out_data));


	

  // finalizing
	 (hipFree(ws_data));
	 (hipFree(out_data));
	 (cudnnDestroyTensorDescriptor(out_desc));
	 (cudnnDestroyConvolutionDescriptor(conv_desc));
	 (hipFree(filt_data));
	 (cudnnDestroyFilterDescriptor(filt_desc));
	 (hipFree(in_data));
	 (cudnnDestroyTensorDescriptor(in_desc));
	 (cudnnDestroy(cudnn));
  
       hipEventRecord(stop, 0);
       hipEventSynchronize(stop);
       hipEventElapsedTime(&elapsedTime1, start, stop);
       cout << elapsedTime1<< endl; //ms
       hipEventDestroy(start);
       hipEventDestroy(stop);
      }
  
  
  return 0;
}
