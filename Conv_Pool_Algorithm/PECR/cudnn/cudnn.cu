#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <stdio.h>
#include <malloc.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <assert.h>



using namespace std;


#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  cudnnStatus_t err = (f); \
  if (err != CUDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}


int main() {
    string a[4995];
	int h[4995],w[4995];
	ifstream file_list("../../dataset/file_list");
	for(int i=0;i<4995;i++)
		file_list>>a[i];
	file_list.close();
	ifstream conv_shape("../../dataset/conv_shape");
	for(int i=0;i<4995;i++){
		conv_shape>>h[i];
		w[i] = h[i];
	}
	conv_shape.close();

for(int i=0;i<4995;i++){
  cudnnHandle_t cudnn;
  CUDNN_CALL(cudnnCreate(&cudnn));

  // input
  const int in_n = 1;
  const int in_c = 1;
  const int in_h = h[i];
  const int in_w = w[i];


  cudnnTensorDescriptor_t in_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&in_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        in_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        in_n, in_c, in_h, in_w));


  float *data = (float*)malloc(in_n * in_c * in_h * in_w * sizeof(float));
  int len = 0;
  ifstream conv_feature(("../../dataset/conv/"+a[i]).c_str());
  while(!conv_feature.eof())
    conv_feature>>data[len++];
  conv_feature.close();

// filter
const int filt_k = 1;
const int filt_c = 1;
const int filt_h = 3;
const int filt_w = 3;



float kernel[9] = { 1,0,1,0,1,0,1,0,0};
float *filt_data;
  float *in_data;



  // convolution
  const int pad_h = 0;
  const int pad_w = 0;
  const int str_h = 1;
  const int str_w = 1;
  const int dil_h = 1;
  const int dil_w = 1;
 // output
 int out_n;
 int out_c;
 int out_h;
 int out_w;

 float *out_data;

 hipEvent_t start, stop;
 float elapsedTime1 = 0.0;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventRecord(start,0);

  CUDA_CALL(hipMalloc(
        &in_data, in_n * in_c * in_h * in_w * sizeof(float)));


  


  cudnnFilterDescriptor_t filt_desc;
  CUDNN_CALL(cudnnCreateFilterDescriptor(&filt_desc));
  CUDNN_CALL(cudnnSetFilter4dDescriptor(
        filt_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));


  CUDA_CALL(hipMalloc(
      &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));




  cudnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(cudnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        CUDNN_CONVOLUTION, CUDNN_DATA_FLOAT));



  CUDNN_CALL(cudnnGetConvolution2dForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        &out_n, &out_c, &out_h, &out_w));



  cudnnTensorDescriptor_t out_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        out_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

 
  CUDA_CALL(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

  // algorithm
  const int n_requestedAlgo = 10;
  cudnnConvolutionFwdAlgoPerf_t algo_perf[n_requestedAlgo];
  int n_returnedAlgo;

  CUDNN_CALL(cudnnFindConvolutionForwardAlgorithm(
      cudnn,
      in_desc, filt_desc, conv_desc, out_desc,
      n_requestedAlgo, &n_returnedAlgo, algo_perf));

  cudnnConvolutionFwdAlgo_t algo = algo_perf[0].algo;


  // workspace
  size_t ws_size;
  CUDNN_CALL(cudnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

  float *ws_data;
  CUDA_CALL(hipMalloc(&ws_data, ws_size));

  float * con_result = (float*)malloc(out_n * out_c * out_h * out_w *sizeof(float));
  // perform
  float alpha = 1.f;
  float beta = 0.f;



 hipMemcpy(filt_data,kernel,filt_k * filt_c * filt_h * filt_w *sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(in_data,data,in_n * in_c * in_h * in_w *sizeof(float),hipMemcpyHostToDevice);


  CUDNN_CALL(cudnnConvolutionForward(
      cudnn,
      &alpha, in_desc, in_data, filt_desc, filt_data,
      conv_desc, algo, ws_data, ws_size,
      &beta, out_desc, out_data));

  // results

  CUDA_CALL(hipMemcpy(
        con_result, out_data,
        out_n * out_c * out_h * out_w * sizeof(float),
        hipMemcpyDeviceToHost));


  // finalizing
  CUDA_CALL(hipFree(ws_data));
  CUDA_CALL(hipFree(out_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(out_desc));
  CUDNN_CALL(cudnnDestroyConvolutionDescriptor(conv_desc));
  CUDA_CALL(hipFree(filt_data));
  CUDNN_CALL(cudnnDestroyFilterDescriptor(filt_desc));
  CUDA_CALL(hipFree(in_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(in_desc));
  CUDNN_CALL(cudnnDestroy(cudnn));


  cudnnHandle_t cudnn_p;
  cudnnPoolingDescriptor_t pooling_desc;
  CUDNN_CALL(cudnnCreatePoolingDescriptor(&pooling_desc));
  cudnnSetPooling2dDescriptor(pooling_desc,            //descriptor handle
                                         CUDNN_POOLING_MAX,       //mode - max pooling
                                         CUDNN_NOT_PROPAGATE_NAN, //NaN propagation mode
                                         2,                       //window height
                                         2,                       //window width
                                         0,                       //vertical padding
                                         0,                       //horizontal padding
                                         1,                       //vertical stride
                                         1);
  cudnnTensorDescriptor_t in_p_desc;
  cudnnCreateTensorDescriptor(&in_p_desc);
  cudnnSetTensor4dDescriptor(in_p_desc,                  //descriptor handle
                                        CUDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DATA_FLOAT,              //data type (precision)
                                        out_n,                        //number of images
                                        out_c,                        //number of channels
                                        out_h,                       //data height
                                        out_w);

  cudnnTensorDescriptor_t out_p_desc;
  cudnnCreateTensorDescriptor(&out_p_desc);
  cudnnSetTensor4dDescriptor(out_p_desc,                 //descriptor handle
                                        CUDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DATA_FLOAT,              //data type (precision)
                                        1,                        //number of images
                                        1,                        //number of channels
                                        out_h-2 +1,                        //data height
                                        out_w-2 +1);
  float a = 1.0f;
  float be = 0.0f;

  float *gpu_in;
  float *gpu_out;

  hipMalloc(&gpu_in,out_n * out_c * out_h * out_w *sizeof(float));
  hipMalloc(&gpu_out, (out_h-2 +1)*(out_h-2 +1)*sizeof(float));
    hipMemset(out_data,0,(out_h-2 +1)*(out_h-2 +1)*sizeof(float));
    hipMemset(out_data,0,(out_h-2 +1)*(out_h-2 +1)*sizeof(float));

  hipMemcpy(gpu_in,con_result,out_n * out_c * out_h * out_w *sizeof(float),hipMemcpyHostToDevice);



  cudnnPoolingForward(cudnn_p,         //cuDNN context handle
                                 pooling_desc,  //pooling descriptor handle
                                 &a,        //alpha scaling factor
                                 in_p_desc,       //input tensor descriptor
                                 gpu_in,       //input data pointer to GPU memory
                                 &be,         //beta scaling factor
                                 out_p_desc,      //output tensor descriptor
                                 gpu_out);



  cudnnDestroyTensorDescriptor(in_p_desc);
  cudnnDestroyTensorDescriptor(out_p_desc);
  cudnnDestroyPoolingDescriptor(pooling_desc);
  cudnnDestroy(cudnn_p);

  hipFree(gpu_in);
  hipFree(gpu_out);

  free(con_result);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime1, start, stop);
  cout<<elapsedTime1<<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);
  }
  return 0;
}
